#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <omp.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

// setting the number of threads:
// #ifndef NUMT
// #define NUMT		2
// #endif

#ifndef BLOCKSIZE
#define BLOCKSIZE		64		// number of threads per block
#endif

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	1000
#endif

// how many tries to discover the maximum performance:
#ifndef NUMTRIES
#define NUMTRIES	10
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

// ranges for the random numbers:
const float XCMIN =	 0.0;
const float XCMAX =	 2.0;
const float YCMIN =	 0.0;
const float YCMAX =	 2.0;
const float RMIN  =	 0.5;
const float RMAX  =	 2.0;

// function prototypes:
float		Ranf( float, float );
int			Ranf( int, int );
void		TimeOfDaySeed( );


// array multiplication (CUDA Kernel) on the device: C = A * B

__global__  void MonteCarlo( float *A, float *B, float *C )
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;

	prods[tnum] = A[gid] * B[gid];
	int numHits = 0;
	for( int t = 0; t < NUMTRIES; t++ )
        {
			for( int n = 0; n < NUMTRIALS; n++ )
			{
				// randomize the location and radius of the circle:
				// float xc = xcs[n];
				// float yc = ycs[n];
				// float  r =  rs[n];

				float xc = A[n];
				float yc = B[n];
				float  r =  C[n];

				// solve for the intersection using the quadratic formula:
				float a = 2.;
				float b = -2.*( xc + yc );
				float c = xc*xc + yc*yc - r*r;
				float d = b*b - 4.*a*c;

				// If d is less than 0., then the circle was completely missed. (Case A) Continue on to the next trial in the for-loop.
				if (d < 0.)
				{
					continue;
				}

				// hits the circle:
				// get the first intersection:
				d = sqrt( d );
				float t1 = (-b + d ) / ( 2.*a );	// time to intersect the circle
				float t2 = (-b - d ) / ( 2.*a );	// time to intersect the circle
				float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection
				
				// If tmin is less than 0., then the circle completely engulfs the laser pointer. (Case B) Continue on to the next trial in the for-loop.
				if (tmin < 0.)
				{
					continue;
				}

				// where does it intersect the circle?
				float xcir = tmin;
				float ycir = tmin;

				// get the unitized normal vector at the point of intersection:
				float nx = xcir - xc;
				float ny = ycir - yc;
				float n = sqrt( nx*nx + ny*ny );
				nx /= n;	// unit vector
				ny /= n;	// unit vector

				// get the unitized incoming vector:
				float inx = xcir - 0.;
				float iny = ycir - 0.;
				float in = sqrt( inx*inx + iny*iny );
				inx /= in;	// unit vector
				iny /= in;	// unit vector

				// get the outgoing (bounced) vector:
				float dot = inx*nx + iny*ny;
				float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
				float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

				// find out if it hits the infinite plate:
				float t = ( 0. - ycir ) / outy;
				// If t is less than 0., then the reflected beam went up instead of down. Continue on to the next trial in the for-loop.

				if(t < 0.)
				{
					continue;
				}
				// Otherwise, this beam hit the infinite plate. (Case D) Increment the number of hits and continue on to the next trial in the for-loop.
				numHits++;
			}
		}


	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		C[wgNum] = prods[0];
}


// main program:

int
main( int argc, char* argv[ ] )
{
	TimeOfDaySeed( );		// seed the random number generator

	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float * hA = new float [ SIZE ];
	float * hB = new float [ SIZE ];
	float * hC = new float [ SIZE/BLOCKSIZE ];

	// From OpenCL MonteCarlo
	// better to define these here so that the rand() calls don't get into the thread timing:
	float *xcs = new float [NUMTRIALS];
	float *ycs = new float [NUMTRIALS];
	float * rs = new float [NUMTRIALS];

	// fill the random-value arrays:
	for( int n = 0; n < NUMTRIALS; n++ )
	{       
			xcs[n] = Ranf( XCMIN, XCMAX );
			ycs[n] = Ranf( YCMIN, YCMAX );
			rs[n] = Ranf(  RMIN,  RMAX ); 
	}    

	for( int i = 0; i < SIZE; i++ )
	{
		hA[i] = hB[i] = (float) sqrt(  (float)(i+1)  );
	}

	// allocate device memory:

	float *dA, *dB, *dC;

	dim3 dimsA( SIZE, 1, 1 );
	dim3 dimsB( SIZE, 1, 1 );
	dim3 dimsC( SIZE/BLOCKSIZE, 1, 1 );

	float *dXCS, *dYCS, *dRS;

	dim3 dimsXCS( NUMTRIALS, 1, 1 );
	dim3 dimsYCS( NUMTRIALS, 1, 1 );
	dim3 dimsRS( NUMTRIALS/BLOCKSIZE, 1, 1 );

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dA), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dB), SIZE*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dC), (SIZE/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );
	
	// hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dXCS), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dYCS), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dRS), (NUMTRIALS/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dA, hA, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dB, hB, SIZE*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	status = hipMemcpy( dXCS, xcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );
	status = hipMemcpy( dYCS, ycs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
	checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( SIZE / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
			// ArrayMul<<< grid, threads >>>( dA, dB, dC );
			MonteCarlo<<< grid, threads >>>( dXCS, dYCS, dRS );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaMultTrials/Second = %10.2lf\n", SIZE, megaMultsPerSecond );

	fprintf( stderr, "BLOCKSIZE Size = %d, NUMTRIALS = %d\n", BLOCKSIZE, NUMTRIALS );

	// copy result from the device to the host:

	status = hipMemcpy( hC, dC, (SIZE/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
		checkCudaErrors( status );
	
	status = hipMemcpy( rs, dRS, (NUMTRIALS/BLOCKSIZE)*sizeof(float), hipMemcpyDeviceToHost );
	checkCudaErrors( status );

	// check the sum :

	// double sum = 0.;
	// for(int i = 0; i < SIZE/BLOCKSIZE; i++ )
	// {
	// 	//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
	// 	sum += (double)hC[i];
	// }
	// fprintf( stderr, "\nsum = %10.2lf\n", sum );

	int numHits = 0;
	for(int i = 0; i < NUMTRIALS/BLOCKSIZE; i++ )
	{
		//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
		numHits += (int)rs[i];
	}
	fprintf( stderr, "\nnumHits = %d\n", numHits );

	//Print Execution Time

	// clean up memory:
	delete [ ] hA;
	delete [ ] hB;
	delete [ ] hC;

	delete [ ] xcs;
	delete [ ] ycs;
	delete [ ] rs;

	status = hipFree( dA );
		checkCudaErrors( status );
	status = hipFree( dB );
		checkCudaErrors( status );
	status = hipFree( dC );
		checkCudaErrors( status );

	status = hipFree( dXCS );
		checkCudaErrors( status );
	status = hipFree( dYCS );
		checkCudaErrors( status );
	status = hipFree( dRS );
		checkCudaErrors( status );


	return 0;
}




//Helper Functions
float
Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}
